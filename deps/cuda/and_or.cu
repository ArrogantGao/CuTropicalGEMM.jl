#include <stdio.h>
#include <stdlib.h>

#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <vector>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>


#define OFFSET(row, col, ld) ((row) * (ld) + (col))


#define OPERATOR_ADD(a, b) (a || b)
#define OPERATOR_MUT(a, b) (a && b)
#define PADDING false

template <
    const int BLOCK_SIZE_M,  // width of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // height of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // height of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X
    > 
__global__ void Bool_andor_kernel( 
    bool * __restrict__ A,
    bool * __restrict__ B,
    bool * __restrict__ C, 
    const int M,
    const int N,
    const int K
    ) {
    
    // size of thread block
    const int bszx = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int bszy = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = bszy * bszx;

    // thread id
    const int tid = threadIdx.y * bszx + threadIdx.x;

    // shared memory

    __shared__ bool As[BLOCK_SIZE_M][BLOCK_SIZE_K]; // avoid bank conflict
    __shared__ bool Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    bool accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {PADDING};
    
    // row number and col number that needs to be loaded blockIdx.y this thread
    const int A_TILE_ROW = tid / BLOCK_SIZE_K;
    const int B_TILE_ROW = tid / BLOCK_SIZE_N;

    const int A_TILE_COL = tid % BLOCK_SIZE_K;
    const int B_TILE_COL = tid % BLOCK_SIZE_N;
    
    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / BLOCK_SIZE_K;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / BLOCK_SIZE_N;

    const int A_S = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int B_S = BLOCK_SIZE_N / THREAD_SIZE_X;

    // can not unroll since K can not be determined at this point
    for (int tile_idx = 0 ; tile_idx < K ; tile_idx += BLOCK_SIZE_K) {

        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
            const int row = BLOCK_SIZE_M * blockIdx.y + i + A_TILE_ROW ;
            const int col = A_TILE_COL + tile_idx;
            if (tile_idx > K - BLOCK_SIZE_K || blockIdx.y == gridDim.y - 1) {
                As[i + A_TILE_ROW ][A_TILE_COL] = row < M && col < K ? A[OFFSET(
                    row, // row
                    col, // col
                    K )] : PADDING;
            } else {
                As[i + A_TILE_ROW ][A_TILE_COL] = A[OFFSET(
                    row, // row
                    col, // col
                    K )];
            }
        }

        // load B from global memory to shared memory
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
            const int row = tile_idx + i + B_TILE_ROW;
            const int col = B_TILE_COL + BLOCK_SIZE_N * blockIdx.x;
            if (blockIdx.x == gridDim.x -1 || tile_idx > K - BLOCK_SIZE_K) {
                Bs[i + B_TILE_ROW][B_TILE_COL] = row < K && col < N ? B[OFFSET(
                    row, // row
                    col, // col
                    N )] : PADDING;
            } else {
                Bs[i + B_TILE_ROW][B_TILE_COL] = B[OFFSET(
                    row, // row
                    col, // col
                    N )];
            }
        }

        __syncthreads();

        // compute c
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE_K; ++ k) {
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] = OPERATOR_ADD(OPERATOR_MUT(As[thread_y * A_S + threadIdx.y][k], Bs[k][thread_x * B_S + threadIdx.x]), accum[thread_y][thread_x]);
                }
            }
            
        }
        __syncthreads();
    }

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
            const int row = BLOCK_SIZE_M * blockIdx.y + thread_y * A_S + threadIdx.y;
            const int col = BLOCK_SIZE_N * blockIdx.x + thread_x * B_S + threadIdx.x;
            if (blockIdx.x == gridDim.x -1 || blockIdx.y == gridDim.y - 1) {
                if (row < M && col < N) {
                    C[OFFSET(row, col, N)] = OPERATOR_ADD(C[OFFSET(row, col, N)], accum[thread_y][thread_x]);
                }
            } else {
                C[OFFSET(row, col, N)] = OPERATOR_ADD(C[OFFSET(row, col, N)], accum[thread_y][thread_x]);
            }
        }
    }
}

extern "C"
void BOOL_andor(const int M, const int N, const int K, bool *d_A, bool *d_B, bool *d_C){

    const int BLOCK_SIZE_M = 96;
    const int BLOCK_SIZE_K = 32;
    const int BLOCK_SIZE_N = 64;
    const int THREAD_SIZE_Y = 6;
    const int THREAD_SIZE_X = 4;

    dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
    dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
    if (N % BLOCK_SIZE_N != 0)
        dimGrid.x++;
    if (M % BLOCK_SIZE_M != 0)
        dimGrid.y++;

    Bool_andor_kernel<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);

}