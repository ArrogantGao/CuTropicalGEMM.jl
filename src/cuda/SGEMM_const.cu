// this script is a common SGEMM method for 

#include <stdio.h>
#include <stdlib.h>

#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <vector>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>


// cal offset from row col and ld , in col-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((col) * (ld) + (row))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])


// GEMM for Col-Major matrix
// default of julia is Col-Major and default of C++ is Row-Major
template <
    const int BLOCK_SIZE_M,  // width of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // height of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // height of block of C that each thread block calculate
    const int THREAD_SIZE_M, // height of block of C that each thread calculate
    const int THREAD_SIZE_N,  // width of block of C that each thread calculate
    const int M, // M, N, K are size of the matrix
    const int K,
    const int N,
    const bool ENABLE_DOUBLE_BUFFER // whether enable double buffering or not
    > 
__global__ void MatMul( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C
    ) {
    
    // size of thread block
    const int bszm = BLOCK_SIZE_M / THREAD_SIZE_M;
    const int bszn = BLOCK_SIZE_N / THREAD_SIZE_N;
    const int THREAD_NUM_PER_BLOCK = bszm * bszn;

    const int BLOCK_SIZE_MK = BLOCK_SIZE_M * BLOCK_SIZE_K;
    const int BLOCK_SIZE_KN = BLOCK_SIZE_K * BLOCK_SIZE_N;
    const int THREAD_SIZE_MN = THREAD_SIZE_M * THREAD_SIZE_N;

    // thread id
    const int tid = threadIdx.y * bszm + threadIdx.x;

    // shared memory
    // directly use 1d shared memory to avoid the conflict of col-major and row-major
    __shared__ float As[BLOCK_SIZE_MK]; // avoid bank conflict
    __shared__ float Bs[BLOCK_SIZE_KN];

    // registers for C
    float accum[THREAD_SIZE_MN] = {0};
    
    // row number and col number that needs to be loaded blockIdx.y this thread
    const int A_TILE_COL = tid / BLOCK_SIZE_M;
    const int B_TILE_COL = tid / BLOCK_SIZE_K;

    const int A_TILE_ROW = tid % BLOCK_SIZE_M;
    const int B_TILE_ROW = tid % BLOCK_SIZE_K;
    
    // col stride that thread uses to load multiple rows of a tile
    // how many cols that the threads load in one iteration
    const int A_TILE_COL_STRIDE = THREAD_NUM_PER_BLOCK / BLOCK_SIZE_M;
    const int B_TILE_COL_STRIDE = THREAD_NUM_PER_BLOCK / BLOCK_SIZE_K;

    // number of threads in M and N direction (used when calculating C)
    // const int A_S = BLOCK_SIZE_M / THREAD_SIZE_M;
    // const int B_S = BLOCK_SIZE_N / THREAD_SIZE_N;

    // (can not unroll since K can not be determined at this point) -- now it is determined XD
    #pragma unroll
    for (int tile_idx = 0 ; tile_idx < K ; tile_idx += BLOCK_SIZE_K) {

        // load A from global memory to shared memory
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_K ; i += A_TILE_COL_STRIDE) {
            const int row = BLOCK_SIZE_M * blockIdx.x + A_TILE_ROW ;
            const int col = A_TILE_COL + i + tile_idx;
            if (blockIdx.x == gridDim.x -1 || blockIdx.y == gridDim.y - 1) {
                As[OFFSET(A_TILE_ROW, i + A_TILE_COL, BLOCK_SIZE_M)] = row < M && col < K ? A[OFFSET(row, col, M)] : 0;
            } else {
                As[OFFSET(A_TILE_ROW, i + A_TILE_COL, BLOCK_SIZE_M)] = A[OFFSET(row, col, M)];
            }
        }

        // load B from global memory to shared memory
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_N; i += B_TILE_COL_STRIDE) {
            const int row = tile_idx + B_TILE_ROW;
            const int col = BLOCK_SIZE_N * blockIdx.y + i + B_TILE_COL;
            if (blockIdx.x == gridDim.x -1 || blockIdx.y == gridDim.y - 1) {
                Bs[OFFSET(B_TILE_ROW, i + B_TILE_COL, BLOCK_SIZE_K)] = row < K && col < N ? B[OFFSET(row, col, K)] : 0;
            } else {
                Bs[OFFSET(B_TILE_ROW, i + B_TILE_COL, BLOCK_SIZE_K)] = B[OFFSET(row, col, K)];
            }
        }

        __syncthreads();

        // compute c
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE_K; ++ k) {
            #pragma unroll
            for (int thread_m = 0; thread_m < THREAD_SIZE_M; ++thread_m) {
                #pragma unroll
                for (int thread_n = 0; thread_n < THREAD_SIZE_N; ++thread_n) {
                    accum[OFFSET(thread_m, thread_n, THREAD_SIZE_M)] += As[OFFSET(threadIdx.x * THREAD_SIZE_M + thread_m, k, BLOCK_SIZE_M)] * Bs[OFFSET(k, threadIdx.y * THREAD_SIZE_N + thread_n, BLOCK_SIZE_K)];
                    // accum[OFFSET(thread_m, thread_n, THREAD_SIZE_M)] = max(As[OFFSET(threadIdx.x * THREAD_SIZE_M + thread_m, k, BLOCK_SIZE_M)] + Bs[OFFSET(k, threadIdx.y * THREAD_SIZE_N + thread_n, BLOCK_SIZE_K)], accum[OFFSET(thread_m, thread_n, THREAD_SIZE_M)]);
                }
            }
            
        }
        __syncthreads();
    }

    // store back to C
    #pragma unroll
    for (int thread_n = 0; thread_n < THREAD_SIZE_N; ++thread_n) {
        #pragma unroll
        for (int thread_m = 0; thread_m < THREAD_SIZE_M; ++thread_m) {
            const int col = BLOCK_SIZE_N * blockIdx.y + THREAD_SIZE_N * threadIdx.y + thread_n;
            const int row = BLOCK_SIZE_M * blockIdx.x + THREAD_SIZE_M * threadIdx.x + thread_m;
            if (blockIdx.x == gridDim.x -1 || blockIdx.y == gridDim.y - 1) {
                if (row < M && col < N) {
                    // C[OFFSET(row, col, M)] = max(accum[OFFSET(thread_m, thread_n, THREAD_SIZE_M)], C[OFFSET(row, col, M)]);
                    C[OFFSET(row, col, M)] += accum[OFFSET(thread_m, thread_n, THREAD_SIZE_M)];
                }
            } else {
                // C[OFFSET(row, col, M)] = max(accum[OFFSET(thread_m, thread_n, THREAD_SIZE_M)], C[OFFSET(row, col, M)]);
                C[OFFSET(row, col, M)] += accum[OFFSET(thread_m, thread_n, THREAD_SIZE_M)];
            }
        }
    }
}

void random_init(float *data, size_t size) {
    for (size_t i = 0; i < size; ++i) {
        data[i] = float(rand()) / RAND_MAX;
    }
}

bool check(const float *A,
    const float *B,
    const float *C,
    const float *D,
    int m, int n, int k) {
    for (int i = 0; i < 20; ++i) {
        int a = rand() % m;
        for (int j = 0; j < 20; ++j) {
            int b = rand() % n;
            float sum = 0.f;
            sum = C[OFFSET(a, b, m)];
            for (int p = 0; p < k; ++p) {
                sum +=  A[OFFSET(a, p, m)] * B[OFFSET(p, b, k)];
            }

            if (std::fabs(sum - D[OFFSET(a, b, m)]) / std::fabs(sum) > 1e-5f) {
                printf("C[%d][%d] not match, %f vs %f\n", a, b, sum, D[OFFSET(a, b, m)]);
                return false;
            }
        }
    }

    return true;
}

int main() {
    const int m = 4096;
    const int n = 4096;
    const int k = 4096;
    const int n_iter = 100;

    const int BLOCK_SIZE_M = 64;
    const int BLOCK_SIZE_K = 32;
    const int BLOCK_SIZE_N = 96;
    const int THREAD_SIZE_M = 4;
    const int THREAD_SIZE_N = 6;
    const bool ENABLE_DOUBLE_BUFFER = false;

    dim3 dimBlock(BLOCK_SIZE_M / THREAD_SIZE_M, BLOCK_SIZE_N / THREAD_SIZE_N);
    dim3 dimGrid(m / BLOCK_SIZE_M, n / BLOCK_SIZE_N);
    if (m % BLOCK_SIZE_M != 0)
        dimGrid.x++;
    if (n % BLOCK_SIZE_N != 0)
        dimGrid.y++;

    float *h_A, *h_B, *h_C, *h_D;
    hipHostMalloc(&h_A, m * k * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_B, k * n * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_C, m * n * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_D, m * n * sizeof(float), hipHostMallocDefault);
    random_init(h_A, m * k);
    random_init(h_B, k * n);
    random_init(h_C, m * n);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_C, h_C, m * n * sizeof(float), hipMemcpyDefault);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);


    // warmup
    MatMul<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_M, THREAD_SIZE_N, m, n, k, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C);

    hipMemcpy(h_D, d_C, m * n * sizeof(float), hipMemcpyDefault);
    bool chk = check(h_A, h_B, h_C, h_D, m, n, k);
    printf("Matrix_C check: %s\n", chk ? "OK" : "Failed");

    hipEventRecord(start);
    for (int i = 0; i < n_iter; ++i) {
        MatMul<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_M, THREAD_SIZE_N, m, n, k, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float ms;
    hipEventElapsedTime(&ms, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);

    long workload = n_iter * long(m) * n * k * 2;
    double tflops = (double(workload) / 1e12) / (double(ms) / 1e3);
    printf("SGEMM with const type Matrix size: %f TFLOPS\n", tflops);
    printf("time per iteration: %f ms\n", ms / n_iter);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipHostFree(h_D);
}

// However, it is shown we will not be able to unroll the loop correctly if we want to use this kernel as a C function correctly...
// but in this script, we did not use any float4 type of some thing like that, it will be great to try to write a julia version of it
// extern "C"
// void SGemmMatMul(const int m, const int n, const int k, const int BM, const int BN, const int BK, const int TM, const int TN, float *d_A, float *d_B, float *d_C){

//     dim3 dimBlock(BM / TM, BN / TN);
//     dim3 dimGrid(m / BM, n / BN);
//     if (m % BM != 0)
//         dimGrid.x++;
//     if (n % BN != 0)
//         dimGrid.y++;

//     const bool ENABLE_DOUBLE_BUFFER = false;

//     MatMul<BM, BK, BN, TM, TN, m, n, k, ENABLE_DOUBLE_BUFFER> 
//     <<< dimGrid, dimBlock >>>(d_A, d_B, d_C);

// }